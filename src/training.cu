#include "hip/hip_runtime.h"
#include "training.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <cmath>
#include <ctime>
#include <random>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

__global__ void adam_update_kernel(float* param, const float* grad, float* m, float* v,
                                   float lr, float beta1, float beta2, float eps, 
                                   float beta1_corrected, float beta2_corrected, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Update biased first moment estimate
        m[idx] = beta1 * m[idx] + (1.0f - beta1) * grad[idx];
        
        // Update biased second raw moment estimate
        v[idx] = beta2 * v[idx] + (1.0f - beta2) * grad[idx] * grad[idx];
        
        // Compute bias-corrected first moment estimate
        float m_hat = m[idx] / (1.0f - beta1_corrected);
        
        // Compute bias-corrected second raw moment estimate
        float v_hat = v[idx] / (1.0f - beta2_corrected);
        
        // Update parameters
        param[idx] -= lr * m_hat / (sqrtf(v_hat) + eps);
    }
}

__global__ void zero_gradients_kernel(float* grad, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad[idx] = 0.0f;
    }
}

__global__ void cross_entropy_loss_kernel(const float* logits, const int* labels, float* loss,
                                          int batch_size, int seq_len, int vocab_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_tokens = batch_size * seq_len;
    
    if (idx < total_tokens) {
        int batch_idx = idx / seq_len;
        int seq_idx = idx % seq_len;
        (void)batch_idx; (void)seq_idx; // Suppress unused variable warnings
        
        int logits_offset = idx * vocab_size;
        int label = labels[idx];
        
        if (label >= 0 && label < vocab_size) {  // Ignore padding tokens (negative labels)
            // Find max for numerical stability
            float max_logit = logits[logits_offset];
            for (int i = 1; i < vocab_size; i++) {
                max_logit = fmaxf(max_logit, logits[logits_offset + i]);
            }
            
            // Compute log softmax
            float sum_exp = 0.0f;
            for (int i = 0; i < vocab_size; i++) {
                sum_exp += expf(logits[logits_offset + i] - max_logit);
            }
            float log_sum_exp = logf(sum_exp) + max_logit;
            
            // Cross entropy loss
            loss[idx] = -(logits[logits_offset + label] - log_sum_exp);
        } else {
            loss[idx] = 0.0f;  // Ignore this token
        }
    }
}

__global__ void cross_entropy_gradient_kernel(const float* logits, const int* labels, float* grad_logits,
                                              int batch_size, int seq_len, int vocab_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = batch_size * seq_len * vocab_size;
    
    if (idx < total_elements) {
        int token_idx = idx / vocab_size;
        int vocab_idx = idx % vocab_size;
        
        int batch_idx = token_idx / seq_len;
        int seq_idx = token_idx % seq_len;
        (void)batch_idx; (void)seq_idx; // Suppress unused variable warnings
        
        int label = labels[token_idx];
        
        if (label >= 0 && label < vocab_size) {  // Valid token
            int logits_offset = token_idx * vocab_size;
            
            // Compute softmax for this token
            float max_logit = logits[logits_offset];
            for (int i = 1; i < vocab_size; i++) {
                max_logit = fmaxf(max_logit, logits[logits_offset + i]);
            }
            
            float sum_exp = 0.0f;
            for (int i = 0; i < vocab_size; i++) {
                sum_exp += expf(logits[logits_offset + i] - max_logit);
            }
            
            float softmax = expf(logits[idx] - max_logit) / sum_exp;
            
            // Gradient of cross entropy loss
            if (vocab_idx == label) {
                grad_logits[idx] = softmax - 1.0f;
            } else {
                grad_logits[idx] = softmax;
            }
        } else {
            grad_logits[idx] = 0.0f;  // Ignore this token
        }
    }
}

AdamOptimizer::AdamOptimizer(const std::vector<Tensor*>& parameters, const TrainingConfig& config)
    : config(config), step_count(0) {
    
    // Initialize momentum and velocity tensors
    for (auto* param : parameters) {
        m_tensors.emplace_back(param->get_shape());
        v_tensors.emplace_back(param->get_shape());
        
        m_tensors.back().zero();
        v_tensors.back().zero();
    }
}

void AdamOptimizer::step(const std::vector<Tensor*>& parameters, const std::vector<Tensor>& gradients) {
    step_count++;
    
    float beta1_corrected = powf(config.beta1, step_count);
    float beta2_corrected = powf(config.beta2, step_count);
    float lr = get_lr();
    
    for (size_t i = 0; i < parameters.size(); i++) {
        int size = parameters[i]->get_size();
        int block_size = 256;
        int grid_size = (size + block_size - 1) / block_size;
        
        adam_update_kernel<<<grid_size, block_size>>>(
            parameters[i]->get_data(),
            gradients[i].get_data(),
            m_tensors[i].get_data(),
            v_tensors[i].get_data(),
            lr, config.beta1, config.beta2, config.eps,
            beta1_corrected, beta2_corrected, size
        );
    }
    
    hipDeviceSynchronize();
}

void AdamOptimizer::zero_grad(const std::vector<Tensor*>& parameters) {
    for (auto* param : parameters) {
        int size = param->get_size();
        int block_size = 256;
        int grid_size = (size + block_size - 1) / block_size;
        
        zero_gradients_kernel<<<grid_size, block_size>>>(param->get_data(), size);
    }
    hipDeviceSynchronize();
}

float AdamOptimizer::get_lr() const {
    if (step_count <= config.warmup_steps) {
        // Linear warmup
        return config.learning_rate * static_cast<float>(step_count) / config.warmup_steps;
    } else {
        // Cosine decay (simplified)
        return config.learning_rate * 0.5f * (1.0f + cosf(M_PI * step_count / 10000.0f));
    }
}

Tokenizer::Tokenizer() {
    // Initialize basic vocabulary
    vocab = {"<pad>", "<eos>", "<unk>"};
    token_to_id["<pad>"] = 0;
    token_to_id["<eos>"] = 1;
    token_to_id["<unk>"] = 2;
    
    // Add basic ASCII characters
    for (char c = 32; c < 127; c++) {
        std::string token(1, c);
        vocab.push_back(token);
        token_to_id[token] = vocab.size() - 1;
    }
    
    // Add common words (simplified)
    std::vector<std::string> common_words = {
        "the", "a", "an", "and", "or", "but", "in", "on", "at", "to", "for", "of", "with", "by",
        "i", "you", "he", "she", "it", "we", "they", "me", "him", "her", "us", "them",
        "this", "that", "these", "those", "here", "there", "where", "when", "why", "how",
        "what", "who", "which", "whose", "is", "are", "was", "were", "be", "been", "being",
        "have", "has", "had", "do", "does", "did", "will", "would", "could", "should", "can",
        "hello", "world", "computer", "program", "code", "function", "variable", "class"
    };
    
    for (const auto& word : common_words) {
        if (token_to_id.find(word) == token_to_id.end()) {
            vocab.push_back(word);
            token_to_id[word] = vocab.size() - 1;
        }
    }
}

std::vector<int> Tokenizer::encode(const std::string& text) {
    std::vector<int> tokens;
    std::istringstream iss(text);
    std::string word;
    
    while (iss >> word) {
        // Convert to lowercase
        std::transform(word.begin(), word.end(), word.begin(), ::tolower);
        
        // Remove punctuation (simplified)
        word.erase(std::remove_if(word.begin(), word.end(), ::ispunct), word.end());
        
        if (!word.empty()) {
            auto it = token_to_id.find(word);
            if (it != token_to_id.end()) {
                tokens.push_back(it->second);
            } else {
                tokens.push_back(unk_token_id);  // Unknown token
            }
        }
    }
    
    return tokens;
}

std::string Tokenizer::decode(const std::vector<int>& tokens) {
    std::string text;
    for (int token_id : tokens) {
        if (token_id >= 0 && token_id < static_cast<int>(vocab.size())) {
            if (!text.empty()) text += " ";
            text += vocab[token_id];
        }
    }
    return text;
}

DataLoader::DataLoader(const TrainingConfig& config, const Tokenizer& tokenizer)
    : config(config), current_idx(0) {
}

void DataLoader::load_data(const std::string& data_path) {
    std::ifstream file(data_path);
    if (!file.is_open()) {
        std::cout << "Warning: Could not open " << data_path << std::endl;
        std::cout << "Creating dummy conversational data..." << std::endl;
        // Create dummy conversational data
        for (int i = 0; i < 100; i++) {
            std::vector<int> dummy_seq(config.seq_length);
            for (int j = 0; j < config.seq_length; j++) {
                dummy_seq[j] = rand() % 100 + 3;  // Smaller vocab range
            }
            sequences.push_back(dummy_seq);
        }
        return;
    }
    
    std::cout << "Loading conversational training data from " << data_path << std::endl;
    
    std::string line;
    Tokenizer tokenizer;
    
    while (std::getline(file, line) && sequences.size() < 10000) {  // Limit dataset size
        auto tokens = tokenizer.encode(line);
        if (tokens.size() >= config.seq_length) {
            // Split long sequences
            for (size_t i = 0; i + config.seq_length <= tokens.size(); i += config.seq_length) {
                std::vector<int> seq(tokens.begin() + i, tokens.begin() + i + config.seq_length);
                sequences.push_back(seq);
            }
        } else if (tokens.size() > 10) {  // Only use sequences with reasonable length
            // Pad short sequences
            tokens.resize(config.seq_length, 0);  // Pad with pad token
            sequences.push_back(tokens);
        }
    }
    
    std::cout << "Loaded " << sequences.size() << " sequences." << std::endl;
}

bool DataLoader::get_next_batch(Tensor& input_ids, Tensor& labels) {
    if (current_idx + config.batch_size > sequences.size()) {
        return false;  // No more batches
    }
    
    // Copy data to tensors
    input_ids.to_host();
    labels.to_host();
    
    for (int b = 0; b < config.batch_size; b++) {
        const auto& seq = sequences[current_idx + b];
        
        for (int s = 0; s < config.seq_length; s++) {
            int input_idx = b * config.seq_length + s;
            
            if (s < config.seq_length - 1) {
                input_ids.get_data()[input_idx] = static_cast<float>(seq[s]);
                labels.get_data()[input_idx] = static_cast<float>(seq[s + 1]);
            } else {
                input_ids.get_data()[input_idx] = static_cast<float>(seq[s]);
                labels.get_data()[input_idx] = 1.0f;  // EOS token
            }
        }
    }
    
    input_ids.to_device();
    labels.to_device();
    
    current_idx += config.batch_size;
    return true;
}

int DataLoader::get_num_batches() const {
    return (sequences.size() + config.batch_size - 1) / config.batch_size;
}

void DataLoader::shuffle() {
    std::shuffle(sequences.begin(), sequences.end(), std::mt19937{std::random_device{}()});
    current_idx = 0;
}

Trainer::Trainer(const ModelConfig& model_config, const TrainingConfig& train_config)
    : config(train_config) {
    
    model = std::make_unique<GPTModel>(model_config);
    tokenizer = std::make_unique<Tokenizer>();
    dataloader = std::make_unique<DataLoader>(train_config, *tokenizer);
    
    auto parameters = model->get_all_parameters();
    optimizer = std::make_unique<AdamOptimizer>(parameters, train_config);
    
    // Load training data - check multiple possible paths
    std::string data_file;
    if (!train_config.data_path.empty()) {
        data_file = train_config.data_path + "train.txt";
    } else {
        // Try different paths depending on where program is run from
        std::vector<std::string> possible_paths = {
            "data/train.txt",           // From project root
            "../data/train.txt",        // From build directory
            "../../data/train.txt"      // From nested build directory
        };
        
        data_file = "data/train.txt"; // Default
        for (const auto& path : possible_paths) {
            std::ifstream test_file(path);
            if (test_file.is_open()) {
                data_file = path;
                test_file.close();
                break;
            }
        }
    }
    dataloader->load_data(data_file);
}

void Trainer::train() {
    auto parameters = model->get_all_parameters();
    
    for (int epoch = 0; epoch < config.max_epochs; epoch++) {
        std::cout << "Epoch " << epoch + 1 << "/" << config.max_epochs << std::endl;
        
        dataloader->shuffle();
        float total_loss = 0.0f;
        int num_batches = 0;
        
        Tensor input_ids({config.batch_size, config.seq_length});
        Tensor labels({config.batch_size, config.seq_length});
        Tensor logits({config.batch_size, config.seq_length, tokenizer->get_vocab_size()});
        
        std::cout << "Starting training batches..." << std::endl;
        
        while (dataloader->get_next_batch(input_ids, labels)) {
            std::cout << "Processing batch " << num_batches + 1 << std::endl;
            
            try {
                // Forward pass
                model->forward(input_ids, logits);
                
                // Compute loss
                float batch_loss = compute_loss(logits, labels);
                total_loss += batch_loss;
                num_batches++;
                
                std::cout << "Batch " << num_batches << ", Loss: " << batch_loss << std::endl;
                
                // Skip backward pass for now to avoid complexity
                // Just do a simple forward pass test
                
            } catch (const std::exception& e) {
                std::cerr << "Error in batch " << num_batches + 1 << ": " << e.what() << std::endl;
                break;
            }
            
            // Limit to just a few batches for testing
            if (num_batches >= 3) break;
        }
        
        float avg_loss = total_loss / num_batches;
        std::cout << "Average loss for epoch " << epoch + 1 << ": " << avg_loss << std::endl;
    }
}

float Trainer::compute_loss(const Tensor& logits, const Tensor& labels) {
    int batch_size = logits.shape[0];
    int seq_len = logits.shape[1];
    int vocab_size = logits.shape[2];
    
    Tensor loss_per_token({batch_size, seq_len});
    Tensor labels_int({batch_size, seq_len});
    
    // Convert float labels to int (simplified)
    Tensor labels_copy = labels.clone();
    labels_copy.to_host();
    labels_int.to_host();
    for (int i = 0; i < batch_size * seq_len; i++) {
        reinterpret_cast<int*>(labels_int.get_data())[i] = static_cast<int>(labels_copy.get_data()[i]);
    }
    labels_int.to_device();
    
    int block_size = 256;
    int grid_size = (batch_size * seq_len + block_size - 1) / block_size;
    
    cross_entropy_loss_kernel<<<grid_size, block_size>>>(
        logits.get_data(),
        reinterpret_cast<int*>(labels_int.get_data()),
        loss_per_token.get_data(),
        batch_size, seq_len, vocab_size
    );
    
    // Sum up the losses
    loss_per_token.to_host();
    float total_loss = 0.0f;
    int valid_tokens = 0;
    
    for (int i = 0; i < batch_size * seq_len; i++) {
        int label = static_cast<int>(labels_copy.get_data()[i]);
        if (label >= 0) {  // Valid token
            total_loss += loss_per_token.get_data()[i];
            valid_tokens++;
        }
    }
    
    return valid_tokens > 0 ? total_loss / valid_tokens : 0.0f;
}

void Trainer::compute_loss_gradient(const Tensor& logits, const Tensor& labels, Tensor& grad_logits) {
    int batch_size = logits.shape[0];
    int seq_len = logits.shape[1];
    int vocab_size = logits.shape[2];
    
    Tensor labels_int({batch_size, seq_len});
    
    // Convert float labels to int
    Tensor labels_copy = labels.clone();
    labels_copy.to_host();
    labels_int.to_host();
    for (int i = 0; i < batch_size * seq_len; i++) {
        reinterpret_cast<int*>(labels_int.get_data())[i] = static_cast<int>(labels_copy.get_data()[i]);
    }
    labels_int.to_device();
    
    int block_size = 256;
    int grid_size = (batch_size * seq_len * vocab_size + block_size - 1) / block_size;
    
    cross_entropy_gradient_kernel<<<grid_size, block_size>>>(
        logits.get_data(),
        reinterpret_cast<int*>(labels_int.get_data()),
        grad_logits.get_data(),
        batch_size, seq_len, vocab_size
    );
    
    hipDeviceSynchronize();
}